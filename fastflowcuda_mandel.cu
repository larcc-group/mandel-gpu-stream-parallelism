#include "hip/hip_runtime.h"
/* ***************************************************************************
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License version 2 as
 *  published by the Free Software Foundation.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, write to the Free Software
 *  Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
 *
 *  As a special exception, you may use this file as part of a free software
 *  library without restriction.  Specifically, if other files instantiate
 *  templates or use macros or inline functions from this file, or you compile
 *  this file and link it with other files to produce an executable, this
 *  file does not by itself cause the resulting executable to be covered by
 *  the GNU General Public License.  This exception does not however
 *  invalidate any other reasons why the executable file might be covered by
 *  the GNU General Public License.
 *
 ****************************************************************************
 */

/*

   Author: Marco Aldinucci.
   email:  aldinuc@di.unipi.it
   marco@pisa.quadrics.com
   date :  15/11/97

Modified by:

****************************************************************************
 *  Author: Dalvan Griebler <dalvangriebler@gmail.com>
 *  Author: Dinei Rockenbach <dinei.rockenbach@edu.pucrs.br>
 *
 *  Copyright: GNU General Public License
 *  Description: This program simply computes the mandelbroat set.
 *  File Name: mandel.cpp
 *  Version: 1.0 (25/05/2018)
 *  Compilation Command: make
 ****************************************************************************
*/


#include <stdio.h>
#if !defined(NO_DISPLAY)
#include "marX2.h"
#endif
#include <sys/time.h>
#include <math.h>

#include <iostream>
#include <chrono>
 
#include "cudabase.h"
int block_size = 32;
int total_devices = 1;

int batch_size = 1; 

__global__ void mandel_kernel(int batch, int batch_size, int dim, double init_a, double init_b, double step, int niter, unsigned char *M) {

    int threadIdGlobal = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i_in_batch = floor((double)threadIdGlobal/dim);
    int i = batch * batch_size + i_in_batch; //global i
    int j = threadIdGlobal - i_in_batch*dim;
    if (i < dim && j < dim) {
		double im = init_b + (step * i); 
		double cr; 
		double a = cr = init_a + step * j; 
		double b = im; 
		int k = 0; 
		for (k = 0; k < niter; k++) {
			double a2 = a*a; 
			double b2 = b*b; 
			if ((a2+b2) > 4.0) break; 
			b = 2*a*b+im;
			a = a2-b2+cr;
		}
		M[i_in_batch*dim+j] = (unsigned char)255-((k*255 / niter));
	} 
}

#include <ff/farm.hpp>
using namespace ff; 

#define DIM 800
#define ITERATION 1024

double diff(struct timeval  a,  struct timeval  b) {
    long sec  = (a.tv_sec  - b.tv_sec);
    long usec = (a.tv_usec - b.tv_usec);

    if(usec < 0) {
        --sec;
        usec += 1000000;
    }
    return ((double)(sec*1000)+ (double)usec/1000.0);
}

struct task_t {
    task_t(int batch, unsigned char *M, unsigned char* dev_M, hipStream_t hip_stream) : batch(batch), M(M), dev_M(dev_M), hip_stream(hip_stream){};
    int batch;
    unsigned char* M;
	unsigned char* dev_M;
	hipStream_t hip_stream;
};
class Emitter: public ff_node_t<task_t> {
public:
    int dim;
    Emitter(int dim): dim(dim) {}
    task_t *svc(task_t*) {
    	int batches = ceil((double)dim / batch_size); 
		for(int batch = 0; batch < batches;batch++) {
			CudaSafeCall( hipSetDevice(batch % total_devices) );

			unsigned char * M; 
    		CudaSafeCall( hipHostAlloc((void**)&M, dim * batch_size, hipHostMallocDefault) );

			unsigned char *dev_M; //on device
			CudaSafeCall( hipMalloc((void**)&dev_M, dim * batch_size) );

			hipStream_t hip_stream;
			CudaSafeCall( hipStreamCreate(&hip_stream) );

            ff_send_out(new task_t(batch, M, dev_M, hip_stream));
        }
        return (task_t*)EOS;
    }
};

class Worker: public ff_node_t<task_t> {
public:
    int dim;
    int niter;
    double init_a;
    double init_b;
    double step;
    Worker(int dim, int niter, double init_a, double init_b, double step): dim(dim), niter(niter), init_a(init_a), init_b(init_b), step(step) {}
    task_t* svc(task_t* t) {

	    CudaSafeCall( hipSetDevice(t->batch % total_devices) );

        int threads = block_size;
        int blocks = ceil((double)dim*batch_size / block_size);
        
    	mandel_kernel<<< blocks, threads, 0, t->hip_stream >>>(t->batch, batch_size, dim, init_a, init_b, step, niter, t->dev_M);

    	CudaSafeCall( hipMemcpyAsync(t->M, t->dev_M, dim * batch_size, hipMemcpyDeviceToHost, t->hip_stream) );

        return t;
    }
};

class Collector: public ff_node_t<task_t> {
public:
    int dim;
    Collector(int dim): dim(dim) {};
	task_t* svc(task_t* t) {

	    CudaSafeCall( hipSetDevice(t->batch % total_devices) );
		CudaSafeCall( hipStreamSynchronize(t->hip_stream) );

#if !defined(NO_DISPLAY)
		for(int i = 0; i < batch_size; i++) {
			ShowLine(&t->M[i*dim], dim, t->batch*batch_size+i);
		} 
#endif
        CudaSafeCall( hipHostFree(t->M) );
        CudaSafeCall( hipFree(t->dev_M) );
		CudaSafeCall( hipStreamDestroy(t->hip_stream) );

    	return (task_t*)GO_ON;
	}
};

int main(int argc, char **argv) {
    double init_a=-2.125,init_b=-1.5,range=3.0;
    int dim = DIM, niter = ITERATION;
    // stats
    struct timeval t1,t2;
    int retries=1;
    double avg = 0;
    int n_workers = 1;
    int num_gpus = 0;

    if (argc<6) {
		printf("Usage: %s size niterations retries workers batch_size [num_gpus]\n", argv[0]); 
        printf("    num_gpus : Number of GPUs that should be used. If not informed, use all available GPUs\n\n");
        exit(-1);
    }
    else {
        dim = atoi(argv[1]);
        niter = atoi(argv[2]);
        retries = atoi(argv[3]);
        n_workers = atoi(argv[4]);
		batch_size = atoi(argv[5]);
        if (argc > 6) {
            num_gpus = atoi(argv[6]);
        }
    }
    double * runs = (double *) malloc(retries*sizeof(double));

    double step = range/((double) dim);

#if !defined(NO_DISPLAY)
    SetupXWindows(dim,dim,1,NULL,"Sequential Mandelbroot");
#endif

    printf("bin;size;numiter;time (ms);workers;batch size\n");

	if (!num_gpus) {
		CudaSafeCall( hipGetDeviceCount(&total_devices) );
		if (total_devices < 1) {
			printf("No CUDA-enabled device found");
			return 1;
		}
	} else {
		total_devices = num_gpus;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	block_size = devProp.maxThreadsPerBlock; //max threads per block

    for (int r=0; r<retries; r++) {

        // Start time
        gettimeofday(&t1,NULL);

        std::vector<ff_node*> workers;
        for (int w=0; w<n_workers; w++) {
            workers.push_back(new Worker(dim, niter, init_a, init_b, step));
        }

        ff_farm<> farm;
        farm.add_emitter(new Emitter(dim));
        farm.add_workers(workers);
        farm.add_collector(new Collector(dim));
        int ret = farm.run_and_wait_end();
        if (ret < 0) {
            printf("Error on FastFlow farm: %d\n", ret);
            return -1;
        }

        // Stop time
        gettimeofday(&t2,NULL);

        CudaCheckError();
        
        avg += runs[r] = diff(t2,t1);
		printf("%s (%d GPU);%d;%d;%.2f;%d;%d\n", argv[0], total_devices, dim, niter, runs[r], n_workers, batch_size);
    }
    avg = avg / (double) retries;
    double var = 0;
    for (int r=0; r<retries; r++) {
        var += (runs[r] - avg) * (runs[r] - avg);
    }
    var /= retries;

#if !defined(NO_DISPLAY)
    printf("Average on %d experiments = %f (ms) Std. Dev. %f\n\nPress a key\n",retries,avg,sqrt(var));
    getchar();
    CloseXWindows();
#endif

    return 0;
}
