#include "hip/hip_runtime.h"
/* ***************************************************************************
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License version 2 as
 *  published by the Free Software Foundation.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, write to the Free Software
 *  Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA 02111-1307, USA.
 *
 *  As a special exception, you may use this file as part of a free software
 *  library without restriction.  Specifically, if other files instantiate
 *  templates or use macros or inline functions from this file, or you compile
 *  this file and link it with other files to produce an executable, this
 *  file does not by itself cause the resulting executable to be covered by
 *  the GNU General Public License.  This exception does not however
 *  invalidate any other reasons why the executable file might be covered by
 *  the GNU General Public License.
 *
 ****************************************************************************
 */

/*

   Author: Marco Aldinucci.
   email:  aldinuc@di.unipi.it
   marco@pisa.quadrics.com
   date :  15/11/97

Modified by:

****************************************************************************
 *  Author: Dalvan Griebler <dalvangriebler@gmail.com>
 *  Author: Dinei Rockenbach <dinei.rockenbach@edu.pucrs.br>
 *
 *  Copyright: GNU General Public License
 *  Description: This program simply computes the mandelbroat set.
 *  File Name: mandel.cpp
 *  Version: 1.0 (25/05/2018)
 *  Compilation Command: make
 ****************************************************************************
*/

#include <stdio.h>
#if !defined(NO_DISPLAY)
#include "marX2.h"
#endif
#include <sys/time.h>
#include <math.h>

#include <iostream>
#include <chrono>
#include <cstring>

#include "tbb/tbb.h"

#include "cudabase.h"

__global__ void mandel_kernel(int batch, int batch_size, int dim, double init_a, double init_b, double step, int niter, unsigned char *M) {

    int threadIdGlobal = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i_in_batch = floor((double)threadIdGlobal/dim);
    int i = batch * batch_size + i_in_batch; //global i
    int j = threadIdGlobal - i_in_batch*dim;
    if (i < dim && j < dim) {
		double im = init_b + (step * i); 
		double cr; 
		double a = cr = init_a + step * j; 
		double b = im; 
		int k = 0; 
		for (k = 0; k < niter; k++) {
			double a2 = a*a; 
			double b2 = b*b; 
			if ((a2+b2) > 4.0) break; 
			b = 2*a*b+im;
			a = a2-b2+cr;
		}
		M[i_in_batch*dim+j] = (unsigned char)255-((k*255 / niter));
	} 
}

unsigned int block_size = 32;
int total_devices = 1;

int batch_size = 1; 

#define DIM 800
#define ITERATION 1024

double diffmsec(struct timeval  a,  struct timeval  b) {
    long sec  = (a.tv_sec  - b.tv_sec);
    long usec = (a.tv_usec - b.tv_usec);

    if(usec < 0) {
        --sec;
        usec += 1000000;
    }
    return ((double)(sec*1000)+ (double)usec/1000.0);
}

struct task_t {
    task_t(int batch, unsigned char *M, unsigned char* dev_M, hipStream_t hip_stream) : batch(batch), M(M), dev_M(dev_M), hip_stream(hip_stream){};
    int batch;
    unsigned char *M;
	unsigned char* dev_M;
	hipStream_t hip_stream;
};
class Emitter: public tbb::filter {
public:
    int dim;
    int batches;
    int batch = 0;
    Emitter(int dim, int batches): 
        tbb::filter(tbb::filter::serial), dim(dim), batches(batches) {}
    void *operator()(void *in) {
        while(batch < batches) {
            int b = batch++;

			CudaSafeCall( hipSetDevice(b % total_devices) );

            unsigned char * M; 
    		CudaSafeCall( hipHostAlloc((void**)&M, dim * batch_size, hipHostMallocDefault) );

            unsigned char *dev_M; //on device
			CudaSafeCall( hipMalloc((void**)&dev_M, dim * batch_size) );

			hipStream_t hip_stream;
			CudaSafeCall( hipStreamCreate(&hip_stream) );

            return new task_t(b, M, dev_M, hip_stream);
        }
        return NULL;
    }
};

class Worker: public tbb::filter {
public:
    int dim;
    int niter;
    double init_a;
    double init_b;
    double step;
    Worker(int dim, int niter, double init_a, double init_b, double step):
        tbb::filter(tbb::filter::parallel), dim(dim), niter(niter), init_a(init_a), init_b(init_b), step(step) {}
    void *operator()(void* in_t) {
        task_t* t = (task_t*)in_t;

	    CudaSafeCall( hipSetDevice(t->batch % total_devices) );

        int threads = block_size;
        int blocks = ceil((double)dim*batch_size / block_size);
        
    	mandel_kernel<<< blocks, threads, 0, t->hip_stream >>>(t->batch, batch_size, dim, init_a, init_b, step, niter, t->dev_M);

    	CudaSafeCall( hipMemcpyAsync(t->M, t->dev_M, dim * batch_size, hipMemcpyDeviceToHost, t->hip_stream) );
    	
        return t;
    }
};

class Collector: public tbb::filter {
public:
    int dim;
    Collector(int dim): 
        tbb::filter(tbb::filter::serial), dim(dim) {};
    void *operator()(void* in_t) {
        task_t* t = (task_t*)in_t;

	    CudaSafeCall( hipSetDevice(t->batch % total_devices) );
		CudaSafeCall( hipStreamSynchronize(t->hip_stream) );

#if !defined(NO_DISPLAY)
		for(int i = 0; i < batch_size; i++) {
            ShowLine(&t->M[i*dim], dim, t->batch*batch_size+i);
        }
#endif

        CudaSafeCall( hipHostFree(t->M) );
        CudaSafeCall( hipFree(t->dev_M) );
		CudaSafeCall( hipStreamDestroy(t->hip_stream) );

    	return NULL;
	}
};

int main(int argc, char **argv) {
    double init_a=-2.125,init_b=-1.5,range=3.0;
    int dim = DIM, niter = ITERATION;
    // stats
    struct timeval t1,t2;
    int retries=1;
    double avg = 0;
    int n_workers = 1;
    int num_gpus = 0;

    if (argc<6) {
        printf("Usage: %s size niterations retries workers batch_size [num_gpus]\n", argv[0]);
        printf("    num_gpus : Number of GPUs that should be used. If not informed, use all available GPUs\n\n");
        exit(-1);
    } else {
        dim = atoi(argv[1]);
        niter = atoi(argv[2]);
        retries = atoi(argv[3]);
        n_workers = atoi(argv[4]);
        batch_size = atoi(argv[5]);
        if (argc > 6) {
            num_gpus = atoi(argv[6]);
        }
    }

    double * runs = (double *) malloc(retries*sizeof(double));

    double step = range/((double) dim);

#if !defined(NO_DISPLAY)
    if (!num_gpus) {
        printf("Using all available GPUs\n");
    } else {
        printf("Using %d GPUs\n", num_gpus);
    }

    SetupXWindows(dim,dim,1,NULL,"Streaming Mandelbroot");
#endif

    printf("bin;size;numiter;time (ms);workers;batch size\n");

	if (!num_gpus) {
		CudaSafeCall( hipGetDeviceCount(&total_devices) );
		if (total_devices < 1) {
			printf("No CUDA-enabled device found");
			return 1;
		}
	} else {
		total_devices = num_gpus;
	}
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	block_size = devProp.maxThreadsPerBlock; //max threads per block

    for (int r=0; r<retries; r++) {

        // Start time
        gettimeofday(&t1,NULL);

        tbb::task_scheduler_init init(n_workers);
        Emitter emitter(dim, ceil((double)dim / batch_size));
        Worker worker(dim, niter, init_a, init_b, step);
        Collector collector(dim);


        tbb::pipeline pipe;
        pipe.add_filter(emitter);
        pipe.add_filter(worker);
        pipe.add_filter(collector);
        pipe.run(n_workers*5);

        // Stop time
        gettimeofday(&t2,NULL);
        
        avg += runs[r] = diffmsec(t2,t1);
        printf("%s (%d GPU);%d;%d;%.2f;%d;%d\n", argv[0], total_devices, dim, niter, runs[r], n_workers, batch_size);
    }
    avg = avg / (double) retries;
    double var = 0;
    for (int r=0; r<retries; r++) {
        var += (runs[r] - avg) * (runs[r] - avg);
    }
    var /= retries;

#if !defined(NO_DISPLAY)
    printf("Average on %d experiments = %f (ms) Std. Dev. %f\n\nPress a key\n",retries,avg,sqrt(var));
    getchar();
    CloseXWindows();
#endif

    return 0;
}
